#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <hip/hip_runtime.h>
#include "kmeans.h"
#include "atomic_utils.h"


// Functor to calculate squared Euclidean distance
struct SquaredDistance {
    const double* points;
    const double* centroids;
    int dims;

    SquaredDistance(const double* p, const double* c, int d) : points(p), centroids(c), dims(d) {}

    __device__
    double operator()(int point_idx, int centroid_idx) const {
        double dist = 0.0;
        for (int d = 0; d < dims; ++d) {
            double diff = points[point_idx * dims + d] - centroids[centroid_idx * dims + d];
            dist += diff * diff;
        }
        return dist;
    }
};



// CUDA kernel to assign points to the nearest centroids
__global__ void assign_points_to_centroids(const double* points, const double* centroids, int* labels, int n_points, int n_centroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_points) {
        double min_dist = INFINITY;
        int best_centroid = 0;

        for (int c = 0; c < n_centroids; ++c) {
            double dist = 0.0;
            for (int d = 0; d < dims; ++d) {
                double diff = points[idx * dims + d] - centroids[c * dims + d];
                dist += diff * diff;
            }
            if (dist < min_dist) {
                min_dist = dist;
                best_centroid = c;
            }
        }
        labels[idx] = best_centroid;
    }
}

// CUDA kernel to compute new centroids
__global__ void compute_new_centroids(const double* points, const int* labels, double* centroids, int* counts, int n_points, int n_centroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_points) {
        int centroid_idx = labels[idx];
        for (int d = 0; d < dims; ++d) {
            atomicAdd(&centroids[centroid_idx * dims + d], points[idx * dims + d]);
        }
        atomicAdd(&counts[centroid_idx], 1);
    }
}

// CUDA kernel to normalize centroids
__global__ void normalize_centroids(double* centroids, const int* counts, int n_centroids, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n_centroids) {
        for (int d = 0; d < dims; ++d) {
            centroids[idx * dims + d] /= counts[idx];
        }
    }
}




// Function to run KMeans with CUDA kernels
void kmeans_thrust(int k, int dims, int max_iters, double threshold,
                   const std::vector<std::vector<double>>& data, 
                   std::vector<int>& labels, std::vector<std::vector<double>>& centroids) {

    int n_points = data.size();

    // Copy data to device
    thrust::host_vector<double> h_points(n_points * dims);
    for (int i = 0; i < n_points; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_points[i * dims + d] = data[i][d];
        }
    }
    thrust::device_vector<double> d_points = h_points;

    // Initialize centroids
    initialize_centroids(k, data, centroids, time(0));

    thrust::host_vector<double> h_centroids(k * dims);
    for (int i = 0; i < k; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_centroids[i * dims + d] = centroids[i][d];
        }
    }
    thrust::device_vector<double> d_centroids = h_centroids;

    thrust::device_vector<int> d_labels(n_points);
    thrust::device_vector<int> d_counts(k);

    for (int iter = 0; iter < max_iters; ++iter) {
        int blocks = (n_points + 255) / 256;

        // Assign points to centroids
        assign_points_to_centroids<<<blocks, 256>>>(thrust::raw_pointer_cast(d_points.data()), thrust::raw_pointer_cast(d_centroids.data()), thrust::raw_pointer_cast(d_labels.data()), n_points, k, dims);
        hipDeviceSynchronize();

        // Reset centroids and counts
        thrust::fill(d_centroids.begin(), d_centroids.end(), 0.0);
        thrust::fill(d_counts.begin(), d_counts.end(), 0);

        // Compute new centroids
        compute_new_centroids<<<blocks, 256>>>(thrust::raw_pointer_cast(d_points.data()), thrust::raw_pointer_cast(d_labels.data()), thrust::raw_pointer_cast(d_centroids.data()), thrust::raw_pointer_cast(d_counts.data()), n_points, k, dims);
        hipDeviceSynchronize();

        // Normalize centroids
        int norm_blocks = (k + 255) / 256;
        normalize_centroids<<<norm_blocks, 256>>>(thrust::raw_pointer_cast(d_centroids.data()), thrust::raw_pointer_cast(d_counts.data()), k, dims);
        hipDeviceSynchronize();
    }

    // Copy the final centroids and labels back to the host
    thrust::copy(d_centroids.begin(), d_centroids.end(), h_centroids.begin());
    for (int i = 0; i < k; ++i) {
        for (int d = 0; d < dims; ++d) {
            centroids[i][d] = h_centroids[i * dims + d];
        }
    }
    thrust::copy(d_labels.begin(), d_labels.end(), labels.begin());
}