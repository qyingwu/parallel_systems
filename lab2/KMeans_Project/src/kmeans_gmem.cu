#include "hip/hip_runtime.h"
#include "kmeans.h"
#include "atomic_utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>

#define CHECK_CUDA_ERROR(err) if (err != hipSuccess) { std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; exit(EXIT_FAILURE); }

// Shared memory size for the block
#define SHARED_MEM_SIZE 1024

// CUDA Kernels Implementation
__global__ void assign_points_to_centroids_gmem(const double* d_points, const double* d_centroids, int* d_labels, int num_points, int k, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_points) {
        double min_dist = INFINITY;
        int best_cluster = -1;

        for (int c = 0; c < k; ++c) {
            double dist = 0.0f;
            for (int d = 0; d < dims; ++d) {
                double diff = d_points[idx * dims + d] - d_centroids[c * dims + d];
                dist += diff * diff;
            }
            if (dist < min_dist) {
                min_dist = dist;
                best_cluster = c;
            }
        }
        d_labels[idx] = best_cluster;
    }
}

__global__ void compute_new_centroids_gmem(const double* d_points, const int* d_labels, double* d_centroids, int* d_cluster_sizes, int num_points, int k, int dims) {
    extern __shared__ double sdata[];  // Shared memory for centroids
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    if (idx < num_points) {
        int cluster_id = d_labels[idx];

        // Initialize shared memory
        for (int d = tid; d < k * dims; d += blockDim.x) {
            sdata[d] = 0.0f;
        }
        __syncthreads();

        // Accumulate centroid contributions into shared memory
        for (int d = 0; d < dims; ++d) {
            atomicAdd(&sdata[cluster_id * dims + d], d_points[idx * dims + d]);
        }
        atomicAdd(&d_cluster_sizes[cluster_id], 1);
        __syncthreads();

        // Write shared memory results back to global memory
        for (int d = tid; d < k * dims; d += blockDim.x) {
            atomicAdd(&d_centroids[d], sdata[d]);
        }
    }
}

__global__ void normalize_centroids_gmem(double* d_centroids, const int* d_cluster_sizes, int k, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int centroid_idx = idx / dims;
    int dim = idx % dims;

    if (centroid_idx < k && d_cluster_sizes[centroid_idx] > 0) {
        d_centroids[centroid_idx * dims + dim] /= d_cluster_sizes[centroid_idx];
    }
}

__global__ void compute_change_gmem(double* d_centroids, double* d_old_centroids, double* d_change, int k, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < k * dims) {
        double diff = d_centroids[idx] - d_old_centroids[idx];
        d_change[idx] = diff * diff;
    }
}

// Function for CUDA KMeans using global memory
void kmeans_cuda_gmem(int k, int dims, int max_iters, double threshold, const std::vector<std::vector<double>>& data,
                      std::vector<int>& labels, std::vector<std::vector<double>>& centroids) {

    int num_points = data.size();
    double* d_points;
    double* d_centroids;
    int* d_labels;
    int* d_cluster_sizes;
    double* d_change;

    // Allocate memory on device
    hipMalloc(&d_points, num_points * dims * sizeof(double));
    hipMalloc(&d_centroids, k * dims * sizeof(double));
    hipMalloc(&d_labels, num_points * sizeof(int));
    hipMalloc(&d_cluster_sizes, k * sizeof(int));
    hipMalloc(&d_change, k * dims * sizeof(double));

    // Initialize data on host
    std::vector<double> h_points(num_points * dims);
    std::vector<double> h_centroids(k * dims);
    std::vector<int> h_labels(num_points);
    std::vector<int> h_cluster_sizes(k, 0);

    for (int i = 0; i < num_points; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_points[i * dims + d] = data[i][d];
        }
    }

    // Use the centroids that were passed in
    for (int i = 0; i < k; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_centroids[i * dims + d] = centroids[i][d];
        }
    }

    // Copy data to device
    hipMemcpy(d_points, h_points.data(), num_points * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids.data(), k * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_cluster_sizes, 0, k * sizeof(int));

    double* d_old_centroids;
    hipMalloc(&d_old_centroids, k * dims * sizeof(double));
    hipMemcpy(d_old_centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToDevice);

    int blockSize = 256;
    int numBlocks = (num_points + blockSize - 1) / blockSize;

    for (int iter = 0; iter < max_iters; ++iter) {
        std::cout << "Running iteration " << iter + 1 << std::endl;

        // Assign points to centroids using global memory
        assign_points_to_centroids_gmem<<<numBlocks, blockSize>>>(d_points, d_centroids, d_labels, num_points, k, dims);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Compute new centroids using global memory and shared memory reduction
        hipMemset(d_cluster_sizes, 0, k * sizeof(int));
        compute_new_centroids_gmem<<<numBlocks, blockSize, k * dims * sizeof(double)>>>(d_points, d_labels, d_centroids, d_cluster_sizes, num_points, k, dims);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Normalize centroids
        int total_threads = k * dims;
        normalize_centroids_gmem<<<(total_threads + blockSize - 1) / blockSize, blockSize>>>(d_centroids, d_cluster_sizes, k, dims);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // Convergence check
        hipMemset(d_change, 0, k * dims * sizeof(double));
        compute_change_gmem<<<(total_threads + blockSize - 1) / blockSize, blockSize>>>(d_centroids, d_old_centroids, d_change, k, dims);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        std::vector<double> h_change(k * dims);
        hipMemcpy(h_change.data(), d_change, k * dims * sizeof(double), hipMemcpyDeviceToHost);

        double total_change = 0.0f;
        for (int i = 0; i < k * dims; ++i) {
            total_change += h_change[i];
        }
        total_change = sqrt(total_change);


        if (total_change < threshold) {
            std::cout << "Converged at iteration " << iter + 1 << std::endl;
            break;
        }

        // Copy current centroids to old centroids for the next iteration
        hipMemcpy(d_old_centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToDevice);
    }

    hipMemcpy(h_centroids.data(), d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_labels.data(), d_labels, num_points * sizeof(int), hipMemcpyDeviceToHost);

    // Convert host centroids back to 2D vector
    for (int i = 0; i < k; ++i) {
        centroids[i].resize(dims);
        for (int d = 0; d < dims; ++d) {
            centroids[i][d] = h_centroids[i * dims + d];
        }
    }

    labels.assign(h_labels.begin(), h_labels.end());

    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_labels);
    hipFree(d_cluster_sizes);
    hipFree(d_change);
    hipFree(d_old_centroids);
}
