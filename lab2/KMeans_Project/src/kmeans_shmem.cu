#include "hip/hip_runtime.h"
#include "kmeans.h"
#include "atomic_utils.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <cstdlib>
#include <ctime>

#define CHECK_CUDA_ERROR(err)     if (err != hipSuccess) {         std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;         exit(EXIT_FAILURE);     }


// CUDA Kernels Implementation
__global__ void assign_points_to_centroids(const double* d_points, const double* d_centroids, int* d_labels, int num_points, int k, int dims) {
    extern __shared__ double s_centroids[];// Shared memory for centroids
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    // Load centroids into shared memory
    for (int c = tid; c < k * dims; c += blockDim.x) {
        s_centroids[c] = d_centroids[c];
    }
    __syncthreads(); // Synchronize threads before using shared memory

    if (idx < num_points) {
        double min_dist = INFINITY;
        int best_cluster = -1;

        // Compute the distance to each centroid using shared memory
        for (int c = 0; c < k; ++c) {
            double dist = 0.0f;
            for (int d = 0; d < dims; ++d) {
                double diff = d_points[idx * dims + d] - s_centroids[c * dims + d];
                dist += diff * diff;
            }
            if (dist < min_dist) {
                min_dist = dist;
                best_cluster = c;
            }
        }
        d_labels[idx] = best_cluster;
    }
}

__global__ void compute_new_centroids(const double* d_points, const int* d_labels, double* d_centroids, int* d_cluster_sizes, int num_points, int k, int dims) {
    extern __shared__ double sdata[];  // Shared memory for partial centroids
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Initialize shared memory for centroids
    for (int i = tid; i < k * dims; i += blockDim.x) {
        sdata[i] = 0.0f;
    }
    __syncthreads();

    if (idx < num_points) {
        int cluster_id = d_labels[idx];

        // Accumulate centroid contributions into shared memory
        for (int d = 0; d < dims; ++d) {
            atomicAdd(&sdata[cluster_id * dims + d], d_points[idx * dims + d]);
        }
        atomicAdd(&d_cluster_sizes[cluster_id], 1);  // Global atomic for cluster sizes
    }
    __syncthreads();

    // Write the shared memory results to global memory (one thread per block reduces contention)
    for (int i = tid; i < k * dims; i += blockDim.x) {
        atomicAdd(&d_centroids[i], sdata[i]);
    }
}

// Parallelized normalization of centroids
__global__ void normalize_centroids(double* d_centroids, const int* d_cluster_sizes, int k, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k * dims) {
        int centroid_idx = idx / dims;
        int dim = idx % dims;
        if (d_cluster_sizes[centroid_idx] > 0) {
            d_centroids[centroid_idx * dims + dim] /= d_cluster_sizes[centroid_idx];
        }
    }
}

__global__ void compute_change(double* d_centroids, double* d_old_centroids, double* d_change, int k, int dims) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < k * dims) {
        double diff = d_centroids[idx] - d_old_centroids[idx];
        d_change[idx] = diff * diff;
    }
}

// KMeans with CUDA shared memory
void kmeans_cuda_shmem(int k, int dims, int max_iters, double threshold, const std::vector<std::vector<double>>& data,
                       std::vector<int>& labels, std::vector<std::vector<double>>& centroids) {

    int num_points = data.size();
    double* d_points;
    double* d_centroids;
    int* d_labels;
    int* d_cluster_sizes;
    double* d_change;

    // Allocate memory on device
    hipMalloc(&d_points, num_points * dims * sizeof(double));
    hipMalloc(&d_centroids, k * dims * sizeof(double));
    hipMalloc(&d_labels, num_points * sizeof(int));
    hipMalloc(&d_cluster_sizes, k * sizeof(int));
    hipMalloc(&d_change, k * dims * sizeof(double));

    // Initialize data on host
    std::vector<double> h_points(num_points * dims);
    std::vector<double> h_centroids(k * dims);
    std::vector<int> h_labels(num_points);
    std::vector<int> h_cluster_sizes(k, 0);

    for (int i = 0; i < num_points; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_points[i * dims + d] = data[i][d];
        }
    }

    // Use the centroids passed in from the input
    for (int i = 0; i < k; ++i) {
        for (int d = 0; d < dims; ++d) {
            h_centroids[i * dims + d] = centroids[i][d];
        }
    }

    // Copy data to device
    hipMemcpy(d_points, h_points.data(), num_points * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, h_centroids.data(), k * dims * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_cluster_sizes, 0, k * sizeof(int));

    double* d_old_centroids;
    hipMalloc(&d_old_centroids, k * dims * sizeof(double));
    hipMemcpy(d_old_centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToDevice);

    int blockSize = 256;
    int numBlocks = (num_points + blockSize - 1) / blockSize;
    int sharedMemSize = k * dims * sizeof(double);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);  // Start recording time

    for (int iter = 0; iter < max_iters; ++iter) {
    
        // Assign points to centroids using shared memory
        assign_points_to_centroids<<<numBlocks, blockSize, sharedMemSize>>>(d_points, d_centroids, d_labels, num_points, k, dims);
        CHECK_CUDA_ERROR(hipGetLastError());

        // Reset centroids and cluster sizes before computing new centroids
        hipMemset(d_centroids, 0, k * dims * sizeof(double));
        hipMemset(d_cluster_sizes, 0, k * sizeof(int));
        
        compute_new_centroids<<<numBlocks, blockSize, sharedMemSize>>>(d_points, d_labels, d_centroids, d_cluster_sizes, num_points, k, dims);
        CHECK_CUDA_ERROR(hipGetLastError());

        // Normalize centroids
        int total_threads = k * dims;
        normalize_centroids<<<(total_threads + blockSize - 1) / blockSize, blockSize>>>(d_centroids, d_cluster_sizes, k, dims);
        CHECK_CUDA_ERROR(hipGetLastError());

        // Convergence check
        hipMemset(d_change, 0, k * dims * sizeof(double));
        compute_change<<<(k * dims + blockSize - 1) / blockSize, blockSize>>>(d_centroids, d_old_centroids, d_change, k, dims);
        CHECK_CUDA_ERROR(hipGetLastError());

        std::vector<double> h_change(k * dims);
        hipMemcpy(h_change.data(), d_change, k * dims * sizeof(double), hipMemcpyDeviceToHost);

        // Check per centroid
        bool converged = true;
        for (int i = 0; i < k; ++i) {
            double centroid_change = 0.0f;
            for (int d = 0; d < dims; ++d) {
                centroid_change += h_change[i * dims + d];
            }
            centroid_change = sqrt(centroid_change);

            if (centroid_change > threshold) {
                converged = false;
                break;
            }
        }

        if (converged) {
            std::cout << "Converged at iteration " << iter + 1 << std::endl;
            break;
        }

        // Copy current centroids to old centroids for the next iteration
        hipMemcpy(d_old_centroids, d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToDevice);
    }

    hipEventRecord(stop);  

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time in milliseconds
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Total kernel execution time: " << milliseconds << " ms" << std::endl;

    // Copy final centroids and labels back to host
    hipMemcpy(h_centroids.data(), d_centroids, k * dims * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_labels.data(), d_labels, num_points * sizeof(int), hipMemcpyDeviceToHost);

    // Convert host centroids back to 2D vector
    for (int i = 0; i < k; ++i) {
        centroids[i].resize(dims);
        for (int d = 0; d < dims; ++d) {
            centroids[i][d] = h_centroids[i * dims + d];
        }
    }

    labels.assign(h_labels.begin(), h_labels.end());

    // Free CUDA memory
    hipFree(d_points);
    hipFree(d_centroids);
    hipFree(d_labels);
    hipFree(d_cluster_sizes);
    hipFree(d_change);
    hipFree(d_old_centroids);

    // Destroy the CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
